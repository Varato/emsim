#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include "common.cuh"
#include "SliceBuilder_kernel.h"

// fabsf is cuda-standardard. no need to include.
#define LEFT_CLOSE(a, b) (((b) - (a)) < (1e-12))


__global__
void binAtomsOneSliceKernel(float const atomCoordinates[], unsigned nAtoms,
                               uint32_t const uniqueElemsCount[], unsigned nElems,
                               unsigned n1, unsigned n2, float d1, float d2,
                               float output[])
/*
 * Given atom coordinates within one single slice, bin the coordinates into 2D histograms
 * Logical dimensions of the arrays:
 *     atomCoordinates2d: (nAtoms, 3)
 *     uniqueElems: (nElems, )
 *     uniqueElemsCount: (nElems, )
 *     output: (nElems, n1, n2), and it must be fully filled with zeros.
 *
 *     where nAtoms = reduce_sum(uniqueElemsCount)
 *
 * Here we use 1D grid and 1D block. Each thread finds the two indices for x, y coordiates.
 */
{
    unsigned batch = gridDim.x * blockDim.x;
    unsigned row;

    float start_coord[2] = {-d1 * floorf((float)(n1 + 1)/2.0f), // plus for number of bin edges
                            -d2 * floorf((float)(n2 + 1)/2.0f)};

    float end_coord[2] = {start_coord[0] + (float)n1 * d1,
                          start_coord[1] + (float)n2 * d2};

    int e, accumulatedCount;
    float x, y;
    unsigned i, j;

    unsigned gridFinishedRows = 0;
    while(gridFinishedRows < nAtoms) {
        row = gridFinishedRows + blockIdx.x * blockDim.x + threadIdx.x;
        if (row < nAtoms) {
            accumulatedCount = 0;
            for (e = 0; e < nElems; ++e) {
                accumulatedCount += uniqueElemsCount[e];
                if (row < accumulatedCount) break;
            }
            // assume the slices are along the first dimension, so 1, 2 are x and y coordinates
            x = atomCoordinates[row * 3 + 1];
            y = atomCoordinates[row * 3 + 2];
            if (start_coord[0] <= x && x <= end_coord[0] &&
                start_coord[1] <= y && y <= end_coord[1]) {
                    i = LEFT_CLOSE(x, end_coord[0]) ? n1 - 1 : (unsigned)floorf((x - start_coord[0]) / d1);
                    j = LEFT_CLOSE(y, end_coord[1]) ? n2 - 1 : (unsigned)floorf((y - start_coord[1]) / d2);
                    atomicAdd(output + e*n1*n2 + i*n2 + j, 1.0f);
            }
        }
        gridFinishedRows +=batch;
    }
}


__global__
void binAtomsMultiSlicesKernel(float const atomCoordinates[], unsigned nAtoms,
                    uint32_t const uniqueElemsCount[], unsigned nElems,
                    unsigned n0, unsigned n1, unsigned n2, float d0, float d1, float d2,
                    float output[])
/*
 * Given atom coordinates within one single slice, bin the coordinates into 2D histograms
 * Logical dimensions of the arrays:
 *     atomCoordinates2d: (nAtoms, 3)
 *     uniqueElems: (nElems, )
 *     uniqueElemsCount: (nElems, )
 *     output: (nElems, n0, n1, n2), and it must be fully filled with zeros when passing to the function.
 *
 *     where nAtoms = reduce_sum(uniqueElemsCount)
 *
 * Here we use 1D grid and 1D block. Each thread finds the two indices for x, y, z coordiates.
 */
{
    unsigned batch = gridDim.x * blockDim.x;
    unsigned row;

    unsigned nVox = n0 * n1 * n2;

    float start_coord[3] = {-d0 * floorf((float)(n0 + 1)/2.0f), // plus 1 for number of bin edges
                            -d1 * floorf((float)(n1 + 1)/2.0f),
                            -d2 * floorf((float)(n2 + 1)/2.0f)};
    float end_coord[3] = {start_coord[0] + (float)n0 * d0,
                          start_coord[1] + (float)n1 * d1,
                          start_coord[2] + (float)n2 * d2};

    float x, y, z;
    unsigned i, j, k;
    int e, accumulatedCount;

    unsigned gridFinishedRows = 0;
    while(gridFinishedRows < nAtoms) {
        row = gridFinishedRows + blockIdx.x * blockDim.x + threadIdx.x;
        if (row < nAtoms) {

            accumulatedCount = 0;
            for (e = 0; e < nElems; ++e) {
                accumulatedCount += uniqueElemsCount[e];
                if (row < accumulatedCount) break;
            }
            // assume the slices are along the first dimension, so 1, 2 are x and y coordinates
            x = atomCoordinates[row * 3 + 0];
            y = atomCoordinates[row * 3 + 1];
            z = atomCoordinates[row * 3 + 2];
            if (start_coord[0] <= x && x <= end_coord[0] &&
                start_coord[1] <= y && y <= end_coord[1] &&
                start_coord[2] <= z && z <= end_coord[2]) {
                    i = LEFT_CLOSE(x, end_coord[0]) ? n0 - 1 : (unsigned)floorf((x - start_coord[0]) / d0);
                    j = LEFT_CLOSE(y, end_coord[1]) ? n1 - 1 : (unsigned)floorf((y - start_coord[1]) / d1);
                    k = LEFT_CLOSE(z, end_coord[2]) ? n2 - 1 : (unsigned)floorf((z - start_coord[2]) / d2);
                    atomicAdd(output + e*nVox + i*n1*n2 + j*n2 + k, 1.0f);
            }
        }
        gridFinishedRows +=batch;
    }
}


__global__
void broadcastMulKernel(hipfftComplex *A, hipfftReal const *v, hipfftReal a, unsigned n0, unsigned n1, unsigned n2)
/*
 * A: (n0, n1, n2)
 * v: (n0, n2)
 * In-place computes a*A * v in a broadcasting way so that the result is in shape (n0, n1, n2).
 * A and v must be C-contiguous

 * Thread sliding algorithm:
 *     view v as 1D (n0 * n2, ) vector
 *     view A as 2D (n1, n0 * n2) Matrix  (Notice transpose is needed, but it only affects how we index A)
 *     Each block sliding from up to down and covers all the rows of matrix A.
 *     Because number of threads in each block is limited, we may need multiple blocks to cover all columns.
 */
{
    extern __shared__ hipfftReal s[];

    unsigned nCols = n0 * n2;
    // we use 1D grid and 2D blocks
    unsigned colBatch = gridDim.x * blockDim.y;
    unsigned rowBatch = blockDim.x;
    unsigned col, row;  // to logically index A(n1, n0 * n2), so col also index v(n0*n2, )
    unsigned i, k;      // to logically index A(n0, n1, n2), we don't need j because j = row
    /*
     * The conversion is:
     *     j = row
     *     i = col / n2
     *     k = col % n2 = col - n2 * i
     * The global index of A(n0, n1, n2) is
     *     i * n1 * n2 + j * n2 + k
    */

    unsigned blockStartCol;
    unsigned gridStartCol = 0;
    while (gridStartCol < nCols) {
        blockStartCol = gridStartCol + blockDim.y * blockIdx.x;
        col = blockStartCol + threadIdx.y;

        i = col / n2;
        k = col - n2 * i;

        // the first row in the block copy from v to shared memory
        if (threadIdx.x == 0 && col < nCols) {
            s[threadIdx.y] = v[col];
        }
        __syncthreads();

        // sliding downwords (x direction)
        unsigned block_works_intact = n1 / rowBatch;
        unsigned rows_remained = n1 % rowBatch;

        unsigned start_row = 0;
        if (col < nCols) {
            for (int w = 0; w < block_works_intact; ++w) {
                row = start_row + threadIdx.x;
                A[i*n1*n2 + row*n2 + k].x *= s[threadIdx.y] * a;
                A[i*n1*n2 + row*n2 + k].y *= s[threadIdx.y] * a;
                start_row += rowBatch;
            }

            if (threadIdx.x < rows_remained) {
                row = start_row + threadIdx.x;
                A[i*n1*n2 + row*n2 + k].x *= s[threadIdx.y] * a;
                A[i*n1*n2 + row*n2 + k].y *= s[threadIdx.y] * a;
            }
        }
        gridStartCol += colBatch;
    }
}


__global__
void rowReduceSumKernel(hipfftComplex *A, unsigned n0, unsigned n1, hipfftComplex *output)
/* reduce sum A (n0 by n1) over the first dimension and store the result in rows of output.
 * A must be C-contiguous.
 * blockDim.x must be power of 2.
 *
 * Notes:
 *
 * Run this kernel once cannot achieve the total reduce. Every time a block of 1st dimension blockDim.x
 * finishes computation, blockDim.x * 2 rows in A is reduced to one row.
 *
 * In general, we use a 2D grid which consists of many 2D blocks sliding from top to down and left to right until the whole
 * array is covered. Because every time the 2D grid computes, 2 * gridDim.x * blockDim.x is covered, the number of row direction
 * sliding of the grid is ceil(n0 / (2 * gridDim.x * blockDim.x)).
 * As a result, the array A is reduced to
 *     resultRows = ceil(n0 / (2 * gridDim.x * blockDim.x)) * gridDim.x
 * rows, which is effectively how many blocks compute. The resulted rows are stored in the front of output in C-contiguous mannar.
 * So to totally reduce the array A along its first dimension, do the following:
 *
 *      reduceSum_(A, n0, n1, output);
 *      int resultRows = ceil(n0 / (2 * gridDim.x * blockDim.x)) * gridDim.x;
 *      while (resultRows > 1) {
 *          reduceSum_(output, resultRows, n1, output);
 *          resultRows = ceil(resultRows / (2 * gridDim.x * blockDim.x)) * gridDim.x;
 *      }
 */
{
    extern __shared__ hipfftComplex sA[];  // size is blockDim.x * blockDim.y

    // 2D grid and 2D blocks
    unsigned rowBatch = gridDim.x * blockDim.x * 2;
    unsigned colBatch = gridDim.y * blockDim.y;
    unsigned col;        // to index A(n0, n1)'s coloumn
    unsigned row0, row1; // to index the two rows of A(n0, n1) that's being summed at the first reduction level.

    // slide rightwards
    unsigned blockStartCol;
    unsigned gridStartCol = 0;
    while (gridStartCol < n1) {
        blockStartCol = gridStartCol + blockDim.y * blockIdx.y;
        col = blockStartCol + threadIdx.y;

        // sliding downwards (x direction)
        unsigned girdXSlidingCount = 0;
        unsigned gridStartRow = 0;
        while (gridStartRow < n0) {
            row0 =  gridStartRow + blockIdx.x * blockDim.x * 2 + threadIdx.x;
            row1 = row0 + blockDim.x;
            // copy to the shared memory and do the first level reduction.
            if (col < n1) {
                if (row0 >= n0) {
                    sA[threadIdx.x * blockDim.y + threadIdx.y].x = 0;
                    sA[threadIdx.x * blockDim.y + threadIdx.y].y = 0;


                } else if(row0 < n0 && row1 >= n0) {
                    sA[threadIdx.x * blockDim.y + threadIdx.y].x = A[row0 * n1 + col].x;
                    sA[threadIdx.x * blockDim.y + threadIdx.y].y = A[row0 * n1 + col].y;


                } else {
                    sA[threadIdx.x * blockDim.y + threadIdx.y].x = A[row0 * n1 + col].x + A[row1 * n1 + col].x;
                    sA[threadIdx.x * blockDim.y + threadIdx.y].y = A[row0 * n1 + col].y + A[row1 * n1 + col].y;
                }
            }
            __syncthreads();

            for (unsigned step = blockDim.x / 2; step > 0; step >>= 1u) {
                if (threadIdx.x < step && col < n1) {
                    sA[threadIdx.x * blockDim.y + threadIdx.y].x += sA[(threadIdx.x + step) * blockDim.y + threadIdx.y].x;
                    sA[threadIdx.x * blockDim.y + threadIdx.y].y += sA[(threadIdx.x + step) * blockDim.y + threadIdx.y].y;
                }
                __syncthreads();
            }

            // write result to global memory
            if (threadIdx.x == 0 && col < n1) {
                output[(blockIdx.x + girdXSlidingCount * gridDim.x) * n1 + col].x = sA[threadIdx.y].x;
                output[(blockIdx.x + girdXSlidingCount * gridDim.x) * n1 + col].y = sA[threadIdx.y].y;
            }

            gridStartRow += rowBatch;
            girdXSlidingCount += 1;
        }

        gridStartCol += colBatch;

    }
}


namespace emsim { namespace cuda {

    void binAtomsOneSlice_(float const atomCoordinates[], unsigned nAtoms,
                              uint32_t const uniqueElemsCount[], unsigned nElems,
                              unsigned n1, unsigned n2, float d1, float d2,
                              float output[]) {
        unsigned blockDimX = maxThreadsPerBlock;
        if (blockDimX > nAtoms) blockDimX = nAtoms;
        unsigned gridDimX = (int) ceilf((float) nAtoms / (float) blockDimX);

        binAtomsOneSliceKernel<<<gridDimX, blockDimX>>>(atomCoordinates, nAtoms,
                                                           uniqueElemsCount, nElems,
                                                           n1, n2, d1, d2,
                                                           output);
    }


    void binAtomsMultiSlices_(float const atomCoordinates[], unsigned nAtoms,
                   uint32_t const uniqueElemsCount[], unsigned nElems,
                   unsigned n0, unsigned n1, unsigned n2, float d0, float d1, float d2,
                   float output[]) {
        unsigned blockDimX = maxThreadsPerBlock;
        if (blockDimX > nAtoms) blockDimX = nAtoms;
        unsigned gridDimX = (int) ceilf((float) nAtoms / (float) blockDimX);

        binAtomsMultiSlicesKernel<<<gridDimX, blockDimX>>>(atomCoordinates, nAtoms,
                                                uniqueElemsCount, nElems,
                                                n0, n1, n2, d0, d1, d2,
                                                output);
    }

    void broadCastMul_(hipfftComplex *A_d, hipfftReal *v_d, hipfftReal a, unsigned n0, unsigned n1, unsigned n2) {

        unsigned nCols = n0 * n2;
        unsigned nRows = n1;
        unsigned blockDimX = 1;
        while (blockDimX < nRows && blockDimX <= 32) {
            blockDimX <<= 1u;
        }
        unsigned blockDimY = maxThreadsPerBlock / blockDimX;
        if (nCols < blockDimY) blockDimY = nCols;
        auto gridDimX = (unsigned) ceilf((float) nCols / (float) blockDimY);
        gridDimX = gridDimX > 2147483647 ? 2147483647 : gridDimX;
        dim3 grid(gridDimX);
        dim3 block(blockDimX, blockDimY);
        // printf("grid: (%d,). block: (%d, %d)\n", gridDimX, blockDimX, blockDimY);
        size_t sharedMemSize = sizeof(hipfftReal) * block.y;

        broadcastMulKernel<<<grid, block, sharedMemSize>>>(A_d, v_d, a, n0, n1, n2);
    }


    void rowReduceSum_(hipfftComplex *A_d, unsigned n0, unsigned n1, hipfftComplex *output_d) {

        unsigned nRows = n0;
        hipfftComplex *inputPtr = A_d;
        do {
            // determine block dimensions
            unsigned rowsHalved = (int) ceilf((float) nRows / 2.0f);
            unsigned blockDimX = 1;
            while (blockDimX < rowsHalved && blockDimX <= 8) {
                blockDimX <<= 1u;
            }
            unsigned blockDimY = maxThreadsPerBlock / blockDimX;
            if (n1 < blockDimY) blockDimY = n1;

            // determine grid dimensions
            unsigned gridDimX = (int) ceilf((float) rowsHalved / (float) blockDimX);
            unsigned gridDimY = (int) ceilf((float) n1 / (float) blockDimY);
            gridDimX = gridDimX > 65535 ? 65535 : gridDimX;
            gridDimY = gridDimY > 65535 ? 65535 : gridDimY;
            dim3 grid(gridDimX, gridDimY);
            dim3 block(blockDimX, blockDimY);

            unsigned threadsPerBlock = block.x * block.y;
            size_t sharedMemSize = sizeof(hipfftComplex) * threadsPerBlock;

            rowReduceSumKernel<<<grid, block, sharedMemSize>>>(inputPtr, nRows, n1, output_d);
            nRows = (unsigned) ceilf((float) nRows / (float) (2 * grid.x * block.x)) * grid.x;
        } while (nRows > 1);
    }
} }
