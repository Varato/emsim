//
// Created by Chen on 11/8/2020.
//

#include <hipfft/hipfft.h>
#include <cstdio>

#include "WavePropagator.h"
#include "WavePropagator_kernel.cuh"


namespace emsim { namespace cuda {

    WavePropagator::WavePropagator(int n1, int n2, float pixelSize, float waveLength, float relativityGamma)
            : m_n1(n1), m_n2(n2), m_nPix(n1*n2), m_pixelSize(pixelSize),
              m_waveLength(waveLength), m_relativityGamma(relativityGamma), m_p(0) {

        if (hipfftPlan2d(&m_p, (int) m_n1, (int) m_n2, HIPFFT_C2C) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed\n");
        }
    }

    WavePropagator::~WavePropagator() {
        hipfftDestroy(m_p);
    }

    void WavePropagator::sliceTransmit(hipfftComplex *wave, const hipfftReal *slice, hipfftComplex *waveOut) const {
        waveSliceTransmit(wave, slice, m_nPix, m_waveLength, m_relativityGamma, waveOut);
    }

    void WavePropagator::spacePropagate(hipfftComplex *waveFourier, float dz, hipfftComplex *waveOut) const {
        waveSpacePropagate(waveFourier, m_n1, m_n2, dz, m_waveLength, m_pixelSize, waveOut);
    }

    void WavePropagator::lensPropagate(hipfftComplex *wave, float cs_mm, float defocus, float aperture,
                                       hipfftComplex *waveOut) const {
        if (hipfftExecC2C(m_p, wave, waveOut, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: C2C plan forward executation failed\n");
        }
        waveLensPropagate(waveOut, m_n1, m_n2, m_pixelSize, m_waveLength, cs_mm, defocus, aperture, waveOut);

        if (hipfftExecC2C(m_p, waveOut, waveOut, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: C2C plan forward executation failed\n");
        }
    }

    void WavePropagator::singleSlicePropagate(hipfftComplex *wave, hipfftReal const *slice,
                                              float dz, hipfftComplex *waveOut) const {
        sliceTransmit(wave, slice, waveOut);
        if (hipfftExecC2C(m_p, waveOut, waveOut, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: C2C plan forward executation failed\n");
        }
        spacePropagate(waveOut, dz, waveOut);

        if (hipfftExecC2C(m_p, waveOut, waveOut, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: C2C plan backward executation failed\n");
        }
    }

    void WavePropagator::multiSlicePropagate(hipfftComplex *wave, hipfftReal *multiSlices, unsigned int nSlices, float dz,
                                             hipfftComplex *waveOut) const
    {
        // propagate through the first slice
        singleSlicePropagate(wave, multiSlices, dz, waveOut);
        for (int s = 1; s < nSlices; ++s) {
            singleSlicePropagate(waveOut, multiSlices + s * m_nPix, dz, waveOut);
        }
    }
} }
