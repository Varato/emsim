#include <hip/hip_runtime.h>
//#include <thrust/device_vector.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <math.h>


#include "utils.h"




void build_slices_fourier_cuda_device(float scattering_factors_d[], int n_elems,
                                      float atom_histograms_d[], int n_slices, int n1, int n2,
                                      float output_d[])
/*
 * Logical dimensions of the arrays:
 *     scattering_factors_d: (n_elems, n1, n2 // 2 + 1)
 *     atom_histograms_d:    (n_elems, n_slices, n1, n2)
 *     output_d            : (n_slices, n1, n2)
 * They must be in device memory.
 * 
 * Notice the scattering_factors are halved on their last dimension, because it will be used in c2r FFT transforms.
 */

{
    int n[2] = {n1, n2};
    int n2_half = n2 / 2 + 1;
    int n_pix = n1 * n2;
    int n_pix_half = n1 * n2_half;

    //TODO use thurst vector here
    hipfftComplex* location_phase_d;   // to hold intermediate fft result and do computations on it

    if (hipMalloc((void **)&location_phase_d, sizeof(hipfftComplex) * n_elems * n_slices * n_pix_half) != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }
    
    hipfftHandle p, ip;
    /*
     * hipfftResult hipfftPlanMany(hipfftHandle *plan, int rank, int *n, 
     *                           int *inembed, int istride, int idist, 
     *                           int *onembed, int ostride, int odist, 
     *                           hipfftType type, int batch);
     */
    if (hipfftPlanMany(&p, 2, n,
                      NULL, 1, n_pix, 
                      NULL, 1, n_pix_half,
                      HIPFFT_R2C, n_elems * n_slices) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: Plan creation failed");
    }

    if (hipfftPlanMany(&ip, 2, n,
                      NULL, 1, n_pix_half,
                      NULL, 1, n_pix,
                      HIPFFT_C2R, n_slices) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: Plan creation failed");
    }

    if (hipfftExecR2C(p, (hipfftReal *)atom_histograms_d, location_phase_d) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: R2C plan executation failed");
    }

    broadCastMul(location_phase_d, scattering_factors_d, 1.0f/(float)n_pix, n_elems, n_slices, n_pix_half);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }
    rowReduceSum(location_phase_d, n_elems, n_slices*n_pix_half, location_phase_d);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }

    if (hipfftExecC2R(ip, location_phase_d, output_d) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: C2R plan executation failed");
    }

    hipfftDestroy(p);
    hipfftDestroy(ip);
    hipFree(location_phase_d);
}
