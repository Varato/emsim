#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

__global__ void convolve_fourier(hipfftComplex *location_phase, 
                                 hipfftReal *scattering_factors,
                                 int n_elems, int, n_slices, int n1, int n2,
                                 hipfftComplex *slices_fourier) 
{

}

__host__
int build_slices_fftwf_kernel(float scattering_factors_ifftshifted[], int n_elems,
                              float atom_histograms[], int n_slices, int n1, int n2,
                              float output[])

{
    int n[2] = {n1, n2};
    int n2_half = n2 / 2 + 1;
    int n_pix = n1 * n2;
    int n_pix_half = n1 * n2_half;

    int batch = n_elems * n_slices;
    int full_mem_size = sizeof(float) * n_elems * n_slices * n1 * n2;


    hipfftReal* atom_histograms_device, scattering_factors_device;
    hipfftComplex* location_phase_device;
    hipfftComplex* slices_fourier_device;
    hipMalloc((void **)&atom_histograms_device, sizeof(hipfftReal)*n_elems*n_slices*n_pix);
    hipMalloc((void **)&location_phase_device, sizeof(hipfftComplex)*n_elems*n_slices*n_pix_half);
    hipMalloc((void **)&slices_fourier_device, sizeof(hipfftComplex)*n_elems*n_slices*n_pix_half);
    hipMalloc((void **)&scattering_factors_device, sizeof(hipfftReal)*n_elems*n_pix);

    hipMemcpy(atom_histograms_device, atom_histograms, sizeof(float) * batch * n_pix, hipMemcpyHostToDevice);
    hipMemcpy(scattering_factors_device, scattering_factors_ifftshifted, sizeof(float) * n_elems * n_pix, hipMemcpyHostToDevice);



    hipfftHandle p, ip;
    /*
     * hipfftResult hipfftPlanMany(hipfftHandle *plan, int rank, int *n, 
     *                           int *inembed, int istride, int idist, 
     *                           int *onembed, int ostride, int odist, 
     *                           hipfftType type, int batch);
     */
    hipfftPlanMany(&p, 2, n,
                  NULL, 1, n_pix, 
                  NULL, 1, n_pix_half,
                  HIPFFT_C2R, batch);

    hipfftPlanMany(&ip, 2, n,
                  NULL, 1, n_pix_half,
                  NULL, 1, n_pix
                  HIPFFT_R2C, n_slices);

    hipfftExecR2C(p, (hipfftReal *)atom_histograms_device, location_phase_device);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    build_slices_fftwf_kernel<<<blocksPerGrid,  threadsPerBlock>>>(location_phase_device, 
                                                                   scattering_factors_device, 
                                                                   n_elems, n_slices, n1, n2,
                                                                   slices_fourier_device);

    hipfftExecC2R(ip, slices_fourier, ?);
    hipfftDestroy(p);
    hipfftDestroy(ip);
    hipFree(atom_histograms_device);
    hipFree(location_phase_device);
    hipFree(slices_fourier_device);
    hipFree(scattering_factors_device);
}