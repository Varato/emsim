#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

__global__ void convolve_fourier(hipfftComplex *location_phase, 
                                 hipfftReal *scattering_factors,
                                 int n_elems, int, n_slices, int n1, int n2,
                                 hipfftComplex *slices_fourier) 
{

}

__host__
int build_slices_cufft_kernel(float scattering_factors[], int n_elems,
                              float atom_histograms[], int n_slices, int n1, int n2,
                              float output[])
/*
    Logical dimensions of the input arrays:
        scattering_factors: (n_elems, n1, n2 // 2+1)
        atom_histograms:    (n_elems, n_slices, n1, n2)
    Notice the scattering_factors are halved on theiry last dimension, because it will be used in c2r FFT transforms.
*/

{
    int n[2] = {n1, n2};
    int n2_half = n2 / 2 + 1;
    int n_pix = n1 * n2;
    int n_pix_half = n1 * n2_half;

    int batch = n_elems * n_slices;
    int full_mem_size = sizeof(float) * n_elems * n_slices * n1 * n2;


    hipfftReal* batch_data, scattering_factors_device;
    hipfftComplex* location_phase_device;
    hipfftComplex* slices_fourier_device;
    hipMalloc((void **)&batch_data, sizeof(hipfftReal)*n_elems*n_slices*n_pix);
    hipMalloc((void **)&scattering_factors_device, sizeof(hipfftReal)*n_elems*n_pix);

    hipMalloc((void **)&location_phase_device, sizeof(hipfftComplex)*n_elems*n_slices*n_pix_half);
    hipMalloc((void **)&slices_fourier_device, sizeof(hipfftComplex)*n_elems*n_slices*n_pix_half);

    hipMemcpy(batch_data, atom_histograms, sizeof(float) * batch * n_pix, hipMemcpyHostToDevice);
    hipMemcpy(scattering_factors_device, scattering_factors, sizeof(float) * n_elems * n_pix, hipMemcpyHostToDevice);

    hipfftHandle p, ip;
    /*
     * hipfftResult hipfftPlanMany(hipfftHandle *plan, int rank, int *n, 
     *                           int *inembed, int istride, int idist, 
     *                           int *onembed, int ostride, int odist, 
     *                           hipfftType type, int batch);
     */
    hipfftPlanMany(&p, 2, n,
                  NULL, 1, n_pix, 
                  NULL, 1, n_pix_half,
                  HIPFFT_C2R, batch);

    hipfftPlanMany(&ip, 2, n,
                  NULL, 1, n_pix_half,
                  NULL, 1, n_pix
                  HIPFFT_R2C, n_slices);

    hipfftExecR2C(p, (hipfftReal *)batch_data, location_phase_device);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    build_slices_fftwf_kernel<<<blocksPerGrid,  threadsPerBlock>>>(location_phase_device, 
                                                                   scattering_factors_device, 
                                                                   n_elems, n_slices, n1, n2,
                                                                   slices_fourier_device);

    hipfftExecC2R(ip, slices_fourier, batch_data);
    hipfftDestroy(p);
    hipfftDestroy(ip);
    hipFree(batch_data);
    hipFree(location_phase_device);
    hipFree(slices_fourier_device);
    hipFree(scattering_factors_device);
}