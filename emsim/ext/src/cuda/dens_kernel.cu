#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <math.h>


#include "row_reduce_sum.cuh"
#include "broadcast_mul.cuh"


extern "C" void build_slices_cufft_kernel(float scattering_factors[], int n_elems,
                                          float atom_histograms[], int n_slices, int n1, int n2,
                                          float output[])
/*
    Logical dimensions of the input arrays:
        scattering_factors: (n_elems, n1, n2 // 2 + 1)
        atom_histograms:    (n_elems, n_slices, n1, n2)
    Notice the scattering_factors are halved on theiry last dimension, because it will be used in c2r FFT transforms.
*/

{
    int n[2] = {n1, n2};
    int n2_half = n2 / 2 + 1;
    int n_pix = n1 * n2;
    int n_pix_half = n1 * n2_half;

    int batch = n_elems * n_slices;

    hipfftReal* batch_data_d;          // to hold the atom_histograms and the resulted slices in device memory
    hipfftReal* scattering_factors_d;  // to hold the halved scattering_factors for n_elems elements
    hipfftComplex* location_phase_d;   // to hold intermediate fft result and do computations on it
    if (hipMalloc((void **)&batch_data_d, sizeof(hipfftReal) * batch * n_pix) != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }
    if (hipMalloc((void **)&scattering_factors_d, sizeof(hipfftReal) * n_elems * n_pix_half) != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }
    if (hipMalloc((void **)&location_phase_d, sizeof(hipfftComplex) * batch * n_pix_half) != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }

    if (hipMemcpy(batch_data_d, atom_histograms, sizeof(float) * batch * n_pix, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }
    if (hipMemcpy(scattering_factors_d, scattering_factors, sizeof(float) * n_elems * n_pix_half, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }

    hipfftHandle p, ip;
    /*
     * hipfftResult hipfftPlanMany(hipfftHandle *plan, int rank, int *n, 
     *                           int *inembed, int istride, int idist, 
     *                           int *onembed, int ostride, int odist, 
     *                           hipfftType type, int batch);
     */
    if (hipfftPlanMany(&p, 2, n,
                      NULL, 1, n_pix, 
                      NULL, 1, n_pix_half,
                      HIPFFT_R2C, batch) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: Plan creation failed");
    }

    if (hipfftPlanMany(&ip, 2, n,
                      NULL, 1, n_pix_half,
                      NULL, 1, n_pix,
                      HIPFFT_C2R, n_slices) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: Plan creation failed");
    }

    if (hipfftExecR2C(p, (hipfftReal *)batch_data_d, location_phase_d) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: R2C plan executation failed");
    }

    broadCastMul(location_phase_d, scattering_factors_d, 1.0f/(float)n_pix, n_elems, n_slices, n_pix_half);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }
    rowReduceSum(location_phase_d, n_elems, n_slices*n_pix_half, location_phase_d);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }

    if (hipfftExecC2R(ip, location_phase_d, batch_data_d) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: C2R plan executation failed");
    }


    if (hipMemcpy(output, batch_data_d, sizeof(float)*n_slices*n_pix, hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
    }

    hipfftDestroy(p);
    hipfftDestroy(ip);
    hipFree(batch_data_d);
    hipFree(location_phase_d);
    hipFree(scattering_factors_d);
}
