#include <cstdio>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>

#include "SliceBuilder.h"
#include "SliceBuilder_kernel.h"
#include "utils.h"

namespace emsim {
    SliceBuilder::SliceBuilder(float *scatteringFactors, int nElems,
                               int n1, int n2, float pixelSize)
        : m_scatteringFactors(scatteringFactors), m_nElems(nElems),
          m_pixelSize(pixelSize), m_n1(n1), m_n2(n2), m_p(0), m_ip(0)
    {
        m_n2Half = m_n2 / 2 + 1;
        m_nPix = m_n1 * m_n2;
        m_nPixHalf = m_n1 * m_n2Half;

        int n[2] = {m_n1, m_n2};
        if(hipfftPlanMany(&m_p, 2, n,
                         nullptr, 1, m_nPix,
                         nullptr, 1, m_nPixHalf,
                         HIPFFT_R2C, nElems) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
        }


        if (hipfftPlan2d(&m_ip, m_n1, m_n2, HIPFFT_C2R) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
        }
    }

    SliceBuilder::~SliceBuilder() {
        hipfftDestroy(m_p);
        hipfftDestroy(m_ip);
    }

    void SliceBuilder::sliceGen(float const slcAtomHist[], float output[]) const
    {

        thrust::device_vector<hipfftComplex> locationPhase(m_nElems * m_nPixHalf);
        hipfftComplex* locationPhasePtr = thrust::raw_pointer_cast(&locationPhase[0]);
        if(hipfftExecR2C(m_p, (hipfftReal *)slcAtomHist, locationPhasePtr) != HIPFFT_SUCCESS) {

        }

        broadCastMul(locationPhasePtr, m_scatteringFactors,
                     1.0f/(float)m_nPix, m_nElems, 1, m_nPixHalf);
        if (hipGetLastError() != hipSuccess) {
            fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
        }
        rowReduceSum(locationPhasePtr, m_nElems, m_nPixHalf, locationPhasePtr);
        if (hipGetLastError() != hipSuccess) {
            fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
        }

        if (hipfftExecC2R(m_ip, locationPhasePtr, output) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: C2R plan executation failed");
        }

    }

    void SliceBuilder::binAtomsWithinSlice(float const atomCoordinates[], unsigned nAtoms,
                                           unsigned const uniqueElemsCount[],
                                           float output[]) const
    {

        binAtomsWithinSlice_(atomCoordinates, nAtoms, uniqueElemsCount, m_nElems, m_n1, m_n2, m_pixelSize,output);
    }




    SliceBuilderBatch::SliceBuilderBatch(float *scatteringFactors, int nElems,
                                         int nSlices, int n1, int n2, float pixelSize)
        : m_scatteringFactors(scatteringFactors), m_nElems(nElems),
        m_nSlices(nSlices), m_n1(n1), m_n2(n2), m_pixelSize(pixelSize), m_p(0), m_ip(0)
    {
        m_n2Half = m_n2 / 2 + 1;
        m_nPix = m_n1 * m_n2;
        m_nPixHalf = m_n1 * m_n2Half;

        int n[2] = {m_n1, m_n2};
        if (hipfftPlanMany(&m_p, 2, n,
                          nullptr, 1, m_nPix,
                          nullptr, 1, m_nPixHalf,
                          HIPFFT_R2C, m_nElems * m_nSlices) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
        }

        if (hipfftPlanMany(&m_ip, 2, n,
                          nullptr, 1, m_nPixHalf,
                          nullptr, 1, m_nPix,
                          HIPFFT_C2R, m_nSlices) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
        }

    }


    SliceBuilderBatch::~SliceBuilderBatch() {
        hipfftDestroy(m_p);
        hipfftDestroy(m_ip);
    }

    void SliceBuilderBatch::sliceGenBatch(float *atomHist, int n_slices, float *output) const {

        thrust::device_vector<hipfftComplex> locationPhase(m_nElems * m_nSlices * m_nPixHalf);
        hipfftComplex* locationPhasePtr = thrust::raw_pointer_cast(&locationPhase[0]);

        if (hipfftExecR2C(m_p, (hipfftReal *)atomHist, locationPhasePtr) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: R2C plan executation failed");
        }

        broadCastMul(locationPhasePtr, thrust::raw_pointer_cast(m_scatteringFactors),
                     1.0f/(float)m_nPix, m_nElems, m_nSlices, m_nPixHalf);
        if (hipGetLastError() != hipSuccess) {
            fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
        }
        rowReduceSum(locationPhasePtr, m_nElems, m_nSlices*m_nPixHalf, locationPhasePtr);
        if (hipGetLastError() != hipSuccess) {
            fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
        }

        if (hipfftExecC2R(m_ip, locationPhasePtr, output) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: C2R plan executation failed");
        }
    }
}
