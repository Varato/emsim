#include <cstdio>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>

#include "SliceBuilder.h"
#include "SliceBuilder_kernel.h"
#include "utils.h"

namespace emsim { namespace cuda {
    /*
     * SliceBuilder
     */
    SliceBuilder::SliceBuilder(float *scatteringFactors, int nElems,
                               int n1, int n2, float pixelSize)
        : m_scatteringFactors(scatteringFactors), m_nElems(nElems),
          m_pixelSize(pixelSize), m_n1(n1), m_n2(n2), m_p(0), m_ip(0)
    {
        m_n2Half = m_n2 / 2 + 1;
        m_nPix = m_n1 * m_n2;
        m_nPixHalf = m_n1 * m_n2Half;

        int n[2] = {m_n1, m_n2};
        if(hipfftPlanMany(&m_p, 2, n,
                         nullptr, 1, m_nPix,
                         nullptr, 1, m_nPixHalf,
                         HIPFFT_R2C, nElems) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
        }


        if (hipfftPlan2d(&m_ip, m_n1, m_n2, HIPFFT_C2R) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed");
        }
    }

    SliceBuilder::~SliceBuilder() {
        hipfftDestroy(m_p);
        hipfftDestroy(m_ip);
    }

    void SliceBuilder::sliceGen(float const slcAtomHist[], float output[]) const
    {

        thrust::device_vector<hipfftComplex> locationPhase(m_nElems * m_nPixHalf);
        hipfftComplex* locationPhasePtr = thrust::raw_pointer_cast(&locationPhase[0]);
        if(hipfftExecR2C(m_p, (hipfftReal *)slcAtomHist, locationPhasePtr) != HIPFFT_SUCCESS) {
            fprintf(stderr, "SliceBuilder::sliceGen: CUFFT error: R2C execution failed\n");
        }

        broadCastMul(locationPhasePtr, m_scatteringFactors,
                     1.0f/(float)m_nPix, m_nElems, 1, m_nPixHalf);
        if (hipGetLastError() != hipSuccess) {
            fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
        }
        rowReduceSum(locationPhasePtr, m_nElems, m_nPixHalf, locationPhasePtr);
        if (hipGetLastError() != hipSuccess) {
            fprintf(stderr, "CUDA error: %s", hipGetErrorString(hipGetLastError()));
        }

        if (hipfftExecC2R(m_ip, locationPhasePtr, output) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: C2R plan executation failed");
        }
    }

    void SliceBuilder::binAtomsWithinSlice(float const atomCoordinates[], unsigned nAtoms,
                                           uint32_t const uniqueElemsCount[],
                                           float output[]) const
    {

        binAtomsWithinSlice_(atomCoordinates, nAtoms, uniqueElemsCount, m_nElems, m_n1, m_n2, m_pixelSize,output);
    }


    /*
     * SliceBuilderBatch
     */
    SliceBuilderBatch::SliceBuilderBatch(float *scatteringFactors, int nElems,
                                         int nSlices, int n1, int n2, float dz, float pixelSize)
        : m_scatteringFactors(scatteringFactors), m_nElems(nElems),
          m_nSlices(nSlices), m_n1(n1), m_n2(n2), m_dz(dz), m_pixelSize(pixelSize), m_p(0), m_ip(0)
    {
        m_n2Half = m_n2 / 2 + 1;
        m_nPix = m_n1 * m_n2;
        m_nPixHalf = m_n1 * m_n2Half;

        int n[2] = {m_n1, m_n2};
        if (hipfftPlanMany(&m_p, 2, n,
                          nullptr, 1, m_nPix,
                          nullptr, 1, m_nPixHalf,
                          HIPFFT_R2C, m_nElems * m_nSlices) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed\n");
        }

        if (hipfftPlanMany(&m_ip, 2, n,
                          nullptr, 1, m_nPixHalf,
                          nullptr, 1, m_nPix,
                          HIPFFT_C2R, m_nSlices) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: Plan creation failed\n");
        }
    }


    SliceBuilderBatch::~SliceBuilderBatch() {
        hipfftDestroy(m_p);
        hipfftDestroy(m_ip);
    }

    void SliceBuilderBatch::sliceGenBatch(float *atomHist, float *output) const {

        thrust::device_vector<hipfftComplex> locationPhase(m_nElems * m_nSlices * m_nPixHalf);
        hipfftComplex* locationPhasePtr = thrust::raw_pointer_cast(&locationPhase[0]);

        if ( hipfftExecR2C(m_p, (hipfftReal *)atomHist, locationPhasePtr) != HIPFFT_SUCCESS) {
            fprintf(stderr, "SliceBuilderBatch: CUFFT error: R2C plan executation failed\n");
        }

        broadCastMul(locationPhasePtr, thrust::raw_pointer_cast(m_scatteringFactors),
                     1.0f/(float)m_nPix, m_nElems, m_nSlices, m_nPixHalf);
        if (hipGetLastError() != hipSuccess) {
            fprintf(stderr, "SliceBuilderBatch: CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
        }
        rowReduceSum(locationPhasePtr, m_nElems, m_nSlices*m_nPixHalf, locationPhasePtr);
        if (hipGetLastError() != hipSuccess) {
            fprintf(stderr, "SliceBuilderBatch: CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
        }

        if (hipfftExecC2R(m_ip, locationPhasePtr, output) != HIPFFT_SUCCESS) {
            fprintf(stderr, "SliceBuilderBatch: CUFFT error: C2R plan executation failed\n");
        }
    }

    void SliceBuilderBatch::binAtoms(const float *atomCoordinates, unsigned int nAtoms,
                                     const uint32_t *uniqueElemsCount, float *output) const
    {
        binAtoms_(atomCoordinates, nAtoms,
                  uniqueElemsCount, m_nElems,
                  m_nSlices, m_n1, m_n2,
                  m_dz, m_pixelSize, m_pixelSize,
                  output);
    }
} }
