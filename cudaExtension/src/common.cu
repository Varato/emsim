//
// Created by Chen on 11/8/2020.
//

#include <hip/hip_runtime.h>
#include <cstdio>

int maxThreadsPerblock;

void cudaInit() {
    hipDeviceProp_t prop{};
    if(hipGetDeviceProperties (&prop, 0) != hipSuccess) {
        fprintf(stderr, "CUDA init error: %s\n", hipGetErrorString(hipGetLastError()));
        return;
    }
    maxThreadsPerblock = prop.maxThreadsPerBlock;
}